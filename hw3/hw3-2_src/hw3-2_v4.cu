// testcase 18.1
// -block_size=32 -> 0.72 s
// testcase p12k1
// -block_size=32 -> 12.80 s

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define min(a,b) (a < b ? a : b)
#define ceil(a,b) ((a + b - 1) / b)
#define B 32
#define BLOCK_DIM 32  // CUDA thread block dimension

const int INF = ((1 << 30) - 1);
// const int V = 50010;

int n, m;
static int *Dist;        // Changed to 1D array for better CUDA compatibility
static int *d_Dist;      // Device array

// Phase 1 kernel: Process diagonal block
__global__ void phase1_kernel(int *d_Dist, int n, int round)
{
    // Calculate starting position for this round's block
    int block_start = round * B;
    
    // Calculate actual x,y coordinates this thread will process
    int global_i = block_start + threadIdx.y;
    int global_j = block_start + threadIdx.x;

    if (global_i >= n || global_j >= n) return;
    
    // Process B elements for this block
    int k_end = min(B, n - block_start);
    for (int k = 0; k < k_end; k++)
    {
        d_Dist[global_i * n + global_j] = min(d_Dist[global_i * n + global_j], d_Dist[global_i * n + (block_start + k)] + d_Dist[(block_start + k) * n + global_j]);
        __syncthreads();
    }
}

// Phase 2 kernel: Process row blocks
__global__ void phase2_kernel_row(int *d_Dist, int n, int round)
{
    int block_j = blockIdx.x;
    if (block_j == round) return;  // Skip pivot block
    
    int local_i = threadIdx.y;
    int local_j = threadIdx.x;

    // Calculate global indices
    int pivot_start = round * B;
    int block_j_start = block_j * B;
    int global_i = pivot_start + local_i;
    int global_j = block_j_start + local_j;

    if (global_i >= n || global_j >= n) return;

    int k_end = min(B, n - pivot_start);
    for (int k = 0; k < k_end; k++)
    {
        d_Dist[global_i * n + global_j] = min(d_Dist[global_i * n + global_j], d_Dist[global_i * n + (pivot_start + k)] + d_Dist[(pivot_start + k) * n + global_j]);
        __syncthreads();
    }
}

// Phase 2 kernel: Process column blocks
__global__ void phase2_kernel_col(int *d_Dist, int n, int round)
{
    int block_i = blockIdx.y;
    if (block_i == round) return;  // Skip pivot block
    
    int local_i = threadIdx.y;
    int local_j = threadIdx.x;

    // Calculate global indices
    int pivot_start = round * B;
    int block_i_start = block_i * B;
    int global_i = block_i_start + local_i;
    int global_j = pivot_start + local_j;

    if (global_i >= n || global_j >= n) return;

    int k_end = min(B, n - pivot_start);
    for (int k = 0; k < k_end; k++)
    {
        d_Dist[global_i * n + global_j] = min(d_Dist[global_i * n + global_j], d_Dist[global_i * n + (pivot_start + k)] + d_Dist[(pivot_start + k) * n + global_j]);
        __syncthreads();
    }
}

// Phase 3 kernel: Process remaining blocks
__global__ void phase3_kernel(int *d_Dist, int n, int round)
{
    int block_i = blockIdx.y;
    int block_j = blockIdx.x;
    if (block_i == round && block_j == round) return;  // Skip pivot block
    
    int local_i = threadIdx.y;
    int local_j = threadIdx.x;

    int pivot_start = round * B;
    int block_i_start = block_i * B;
    int block_j_start = block_j * B;
    int global_i = block_i_start + local_i;
    int global_j = block_j_start + local_j;

    if (global_i >= n || global_j >= n) return;

    int min_dist = d_Dist[global_i * n + global_j];
    int k_end = min(B, n - pivot_start);
    for (int k = 0; k < k_end; k++)
    {
        min_dist = min(min_dist, d_Dist[global_i * n + (pivot_start + k)] + d_Dist[(pivot_start + k) * n + global_j]);
    }
    
    d_Dist[global_i * n + global_j] = min_dist;
}

void block_FW()
{
    int round = ceil(n, B);
    dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
    
    for (int r = 0; r < round; ++r)
    {
        // Phase 1: Process diagonal block
        phase1_kernel<<<1, block_dim>>>(d_Dist, n, r);
        
        // Phase 2: Process row and column blocks
        phase2_kernel_row<<<dim3(round, 1), block_dim>>>(d_Dist, n, r);
        phase2_kernel_col<<<dim3(1, round), block_dim>>>(d_Dist, n, r);
        
        // Phase 3: Process remaining blocks
        phase3_kernel<<<dim3(round, round), block_dim>>>(d_Dist, n, r);
    }
}

void input(char* infile)
{
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    
    // Allocate host memory
    Dist = (int*)malloc(n * n * sizeof(int));
    
    // Initialize distances
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            if (i == j)
                Dist[i*n+j] = 0;
            else
                Dist[i*n+j] = INF;
        }
    }

    int pair[3];
    for (int i = 0; i < m; ++i)
    {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0]*n+pair[1]] = pair[2];
    }
    fclose(file);
    
    // Allocate and copy to device memory
    hipMalloc(&d_Dist, n * n * sizeof(int));
    hipMemcpy(d_Dist, Dist, n * n * sizeof(int), hipMemcpyHostToDevice);
}

void output(char* outFileName)
{
    // Copy back to host
    hipMemcpy(Dist, d_Dist, n * n * sizeof(int), hipMemcpyDeviceToHost);
    
    FILE* outfile = fopen(outFileName, "w");
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            Dist[i*n+j] = min(Dist[i*n+j], INF);
        }
        fwrite(&Dist[i*n], sizeof(int), n, outfile);
    }
    fclose(outfile);
    
    // Cleanup
    free(Dist);
    hipFree(d_Dist);
}

// Main function remains mostly the same for now
int main(int argc, char* argv[])
{
    input(argv[1]);
    printf("%d\n", n);
    block_FW();
    output(argv[2]);
    return 0;
}