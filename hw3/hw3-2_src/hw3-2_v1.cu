// 2D h_Dist -V=10000 -> runtime error with pk testcases

#include <hip/hip_runtime.h>
#include <sched.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

#define ceil(a, b) ((a + b - 1) / b)
#define min(a,b) (a < b ? a : b)

#define FW_BLOCK_SIZE 512
#define CUDA_BLOCK_SIZE 32
#define DEV_NO 0
hipDeviceProp_t prop;

const int INF = ((1 << 30) - 1);
const int V = 10000;
int N, M, NUM_THREADS;
static int h_Dist[V][V];
int* d_Dist;

void input(char* infile)
{
    FILE* file = fopen(infile, "rb");
    fread(&N, sizeof(int), 1, file);
    fread(&M, sizeof(int), 1, file);

    #pragma omp parallel for collapse(2) schedule(static)
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            if (i == j)
                h_Dist[i][j] = 0;
            else
                h_Dist[i][j] = INF;
        }
    }

    int pair[3];
    for (int i = 0; i < M; ++i)
    {
        fread(pair, sizeof(int), 3, file);
        h_Dist[pair[0]][pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char* outfile)
{
    FILE* file = fopen(outfile, "w");
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            h_Dist[i][j] = min(h_Dist[i][j], INF);
        }
        fwrite(h_Dist[i], sizeof(int), N, file);
    }
    fclose(file);
}

__global__ void process_block_full(int* d_Dist, int N, int k, int block_internal_start_i, int block_internal_start_j)
{
    int i = block_internal_start_i + blockIdx.y * blockDim.y + threadIdx.y;
    int j = block_internal_start_j + blockIdx.x * blockDim.x + threadIdx.x;
    int ij = i * N + j;
    int ik = i * N + k;
    int kj = k * N + j;
    int new_dist = d_Dist[ik] + d_Dist[kj];
    d_Dist[ij] = min(d_Dist[ij], new_dist);
}

__global__ void process_block_partial(int* d_Dist, int N, int k, int block_internal_start_i, int block_internal_start_j)
{
    int i = block_internal_start_i + blockIdx.y * blockDim.y + threadIdx.y;
    int j = block_internal_start_j + blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < N)
    {
        int ij = i * N + j;
        int ik = i * N + k;
        int kj = k * N + j;
        int new_dist = d_Dist[ik] + d_Dist[kj];
        d_Dist[ij] = min(d_Dist[ij], new_dist);
    }
}

void calculate_region(int round, int block_start_i, int block_start_j, int block_width, int block_height)
{
    int block_end_i = block_start_i + block_height;
    int block_end_j = block_start_j + block_width;

    int k_start = round * FW_BLOCK_SIZE;
    int k_end = min((round + 1) * FW_BLOCK_SIZE, N);

    // #pragma omp parallel for collapse(2) schedule(static)
    for (int b_i = block_start_i; b_i < block_end_i; ++b_i)
    {
        for (int b_j = block_start_j; b_j < block_end_j; ++b_j)
        {
            for (int k = k_start; k < k_end; ++k)
            {
                int block_internal_start_i = b_i * FW_BLOCK_SIZE;
                int block_internal_start_j = b_j * FW_BLOCK_SIZE;

                dim3 blockDim(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);
                dim3 gridDim(ceil(FW_BLOCK_SIZE, CUDA_BLOCK_SIZE), ceil(FW_BLOCK_SIZE, CUDA_BLOCK_SIZE));

                if ((b_i + 1) * FW_BLOCK_SIZE < N && (b_j + 1) * FW_BLOCK_SIZE < N)
                    process_block_full<<<gridDim, blockDim>>>(d_Dist, N, k, block_internal_start_i, block_internal_start_j);
                else
                    process_block_partial<<<gridDim, blockDim>>>(d_Dist, N, k, block_internal_start_i, block_internal_start_j);
            }
        }
    }
}

void blocked_FW()
{
    int round = ceil(N, FW_BLOCK_SIZE);

    for (int r = 0; r < round; ++r)
    {
        /* Phase 1*/
        calculate_region(r, r, r, 1, 1);
        
        /* Phase 2*/
        calculate_region(r, r, 0, r, 1);
        calculate_region(r, r, r + 1, round - r - 1, 1);
        calculate_region(r, 0, r, 1, r);
        calculate_region(r, r + 1, r, 1, round - r - 1);

        /* Phase 3*/
        calculate_region(r, 0, 0, r, r);
        calculate_region(r, 0, r + 1, round - r - 1, r);
        calculate_region(r, r + 1, 0, r, round - r - 1);
        calculate_region(r, r + 1, r + 1, round - r - 1, round - r - 1); 
    }
}

__global__ void normal_fw_kernel(int* d_Dist, int N, int k) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N && j < N)
    {
        int ij = i * N + j;
        int ik = i * N + k;
        int kj = k * N + j;
        int new_dist = d_Dist[ik] + d_Dist[kj];
        d_Dist[ij] = min(d_Dist[ij], new_dist);
    }
}

void normal_FW()
{
    dim3 block(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);
    dim3 grid(ceil(N, CUDA_BLOCK_SIZE), ceil(N, CUDA_BLOCK_SIZE));
    for (int k = 0; k < N; ++k)
    {
        normal_fw_kernel<<<grid, block>>>(d_Dist, N, k);
    }
}

int main(int argc, char* argv[]) {
    cpu_set_t cpu_set;
    sched_getaffinity(0, sizeof(cpu_set), &cpu_set);
    NUM_THREADS = CPU_COUNT(&cpu_set);

    hipSetDevice(DEV_NO);

    input(argv[1]);
    
    // Copy data to GPU
    hipMalloc((void**)&d_Dist, N * N * sizeof(int));
    for (int i = 0; i < N; ++i) {
        hipMemcpy(d_Dist + i * N, &h_Dist[i], N * sizeof(int), hipMemcpyHostToDevice);
    }

    blocked_FW();

    // Copy data back main memory
    for (int i = 0; i < N; ++i) {
        hipMemcpy(&h_Dist[i], d_Dist + i * N, N * sizeof(int), hipMemcpyDeviceToHost);
    }
    output(argv[2]);

    // Free device memory
    hipFree(d_Dist);
    
    return 0;
}