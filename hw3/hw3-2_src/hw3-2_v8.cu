// For profiling and submission
// v6 optimize input + omp    -> (til p31k1) 259.53 s
//                   + unroll ->             249.59 s

#include <hip/hip_runtime.h>

#include <fcntl.h>
#include <nvtx3/nvtx3.hpp>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <unistd.h>

#define min(a,b) (a < b ? a : b)
#define ceil(a,b) ((a + b - 1) / b)
#define FW_BZ 78
#define CUDA_BZ 26
#define WORK_PER_THREAD (FW_BZ / CUDA_BZ)

__device__ const int INF = ((1 << 30) - 1);

int n, m, N;
int* h_Dist;

void input(char* infile)
{
    nvtx3::scoped_range range("Input data");
    // Open file and get size
    int fd = open(infile, O_RDONLY);
    struct stat sb;
    fstat(fd, &sb);
    void* mapped = mmap(NULL, sb.st_size, PROT_READ, MAP_PRIVATE, fd, 0);
    
    // Read N and m
    int* data = (int*)mapped;
    N = data[0];
    m = data[1];
    
    // Calculate padded size
    n = N + FW_BZ - (N % FW_BZ);
    
    // Allocate aligned memory
    h_Dist = (int*)aligned_alloc(32, n * n * sizeof(int));
    
    // Initialize distances using SIMD
    #pragma unroll
    #pragma omp parallel for simd
    for (int i = 0; i < n*n; ++i)
        h_Dist[i] = (i/n == i%n) ? 0 : INF;
    
    // Read edge data efficiently
    int* edges = data + 2;  // Skip N and m
    #pragma unroll
    #pragma omp parallel for
    for (int i = 0; i < m; ++i)
    {
        int src = edges[i*3];
        int dst = edges[i*3 + 1];
        int weight = edges[i*3 + 2];
        h_Dist[src*n + dst] = weight;
    }
    
    munmap(mapped, sb.st_size);
    close(fd);
}

void output(char* outFileName)
{
    nvtx3::scoped_range range("Output data");
    FILE* outfile = fopen(outFileName, "w");

    #pragma unroll
    for (int i = 0; i < N; ++i)
        fwrite(&h_Dist[i * n], sizeof(int), N, outfile);

    fclose(outfile);
    free(h_Dist);
}

// Phase 1 kernel: Process diagonal block
__global__ void phase1_kernel(int *d_Dist, int n, int round)
{
    // Shared memory for the block
    __shared__ int sh_Dist[FW_BZ][FW_BZ];

    const int global_start = round * FW_BZ;

    int local_i, local_j, global_i, global_j;

    // Load data into shared memory
    #pragma unroll
    for (int di = 0; di < WORK_PER_THREAD; di++)
    {
        #pragma unroll
        for (int dj = 0; dj < WORK_PER_THREAD; dj++)
        {
            local_i = threadIdx.y + di * CUDA_BZ;
            local_j = threadIdx.x + dj * CUDA_BZ;
            global_i = global_start + local_i;
            global_j = global_start + local_j;

            sh_Dist[local_i][local_j] = d_Dist[global_i*n+global_j];
        }
    }

    __syncthreads();
    
    // Process FW_BZ elements for this block
    #pragma unroll
    for (int k = 0; k < FW_BZ; k++)
    {
        #pragma unroll
        for(int di = 0; di < WORK_PER_THREAD; di++)
        {
            #pragma unroll
            for(int dj = 0; dj < WORK_PER_THREAD; dj++)
            {
                local_i = threadIdx.y + di * CUDA_BZ;
                local_j = threadIdx.x + dj * CUDA_BZ;

                sh_Dist[local_i][local_j] = min(sh_Dist[local_i][local_j],
                                                sh_Dist[local_i][k] + sh_Dist[k][local_j]);
            }
        }
    }

    // Load data back to global memory
    #pragma unroll
    for (int di = 0; di < WORK_PER_THREAD; di++)
    {
        #pragma unroll
        for (int dj = 0; dj < WORK_PER_THREAD; dj++)
        {
            local_i = threadIdx.y + di * CUDA_BZ;
            local_j = threadIdx.x + dj * CUDA_BZ;
            global_i = global_start + local_i;
            global_j = global_start + local_j;

            d_Dist[global_i*n+global_j] = sh_Dist[local_i][local_j];
        }
    }
}

// Phase 2 kernel: Process row blocks
__global__ void phase2_kernel_row(int *d_Dist, int n, int round)
{
    // Shared memory for the block
    __shared__ int sh_pivot_Dist[FW_BZ][FW_BZ];
    __shared__ int sh_Dist[FW_BZ][FW_BZ];

    const int pivot_start = round * FW_BZ;

    int local_i, local_j, global_i, global_j;
    int p_global_i, p_global_j;

    // Load data into shared memory
    #pragma unroll
    for (int di = 0; di < WORK_PER_THREAD; di++)
    {
        #pragma unroll
        for (int dj = 0; dj < WORK_PER_THREAD; dj++)
        {
            local_i = threadIdx.y + di * CUDA_BZ;
            local_j = threadIdx.x + dj * CUDA_BZ;

            p_global_i = pivot_start + local_i;
            p_global_j = pivot_start + local_j;
            sh_pivot_Dist[local_i][local_j] = d_Dist[p_global_i*n+p_global_j];

            global_i = p_global_i;
            global_j = blockIdx.x * FW_BZ + local_j;
            sh_Dist[local_i][local_j] = d_Dist[global_i*n+global_j];
        }
    }

    __syncthreads();

    // Process FW_BZ elements for this block
    #pragma unroll
    for (int k = 0; k < FW_BZ; k++)
    {
        #pragma unroll
        for(int di = 0; di < WORK_PER_THREAD; di++)
        {
            #pragma unroll
            for(int dj = 0; dj < WORK_PER_THREAD; dj++)
            {
                local_i = threadIdx.y + di * CUDA_BZ;
                local_j = threadIdx.x + dj * CUDA_BZ;

                sh_Dist[local_i][local_j] = min(sh_Dist[local_i][local_j],
                                                sh_pivot_Dist[local_i][k] + sh_Dist[k][local_j]);
            }
        }
    }
    
    // Load data back to global memory
    #pragma unroll
    for (int di = 0; di < WORK_PER_THREAD; di++)
    {
        #pragma unroll
        for (int dj = 0; dj < WORK_PER_THREAD; dj++)
        {
            local_i = threadIdx.y + di * CUDA_BZ;
            local_j = threadIdx.x + dj * CUDA_BZ;

            global_i = pivot_start + local_i;
            global_j = blockIdx.x * FW_BZ + local_j;
            d_Dist[global_i*n+global_j] = sh_Dist[local_i][local_j];
        }
    }
}

// Phase 2 kernel: Process column blocks
__global__ void phase2_kernel_col(int *d_Dist, int n, int round)
{
    // Shared memory for the block
    __shared__ int sh_pivot_Dist[FW_BZ][FW_BZ];
    __shared__ int sh_Dist[FW_BZ][FW_BZ];

    const int pivot_start = round * FW_BZ;

    int local_i, local_j, global_i, global_j;
    int p_global_i, p_global_j;

    // Load data into shared memory
    #pragma unroll
    for (int di = 0; di < WORK_PER_THREAD; di++)
    {
        #pragma unroll
        for (int dj = 0; dj < WORK_PER_THREAD; dj++)
        {
            local_i = threadIdx.y + di * CUDA_BZ;
            local_j = threadIdx.x + dj * CUDA_BZ;

            p_global_i = pivot_start + local_i;
            p_global_j = pivot_start + local_j;
            sh_pivot_Dist[local_i][local_j] = d_Dist[p_global_i*n+p_global_j];

            global_i = blockIdx.x * FW_BZ + local_i;
            global_j = p_global_j;
            sh_Dist[local_i][local_j] = d_Dist[global_i*n+global_j];
        }
    }

    __syncthreads();

    // Process FW_BZ elements for this block
    #pragma unroll
    for (int k = 0; k < FW_BZ; k++)
    {
        #pragma unroll
        for(int di = 0; di < WORK_PER_THREAD; di++)
        {
            #pragma unroll
            for(int dj = 0; dj < WORK_PER_THREAD; dj++)
            {
                local_i = threadIdx.y + di * CUDA_BZ;
                local_j = threadIdx.x + dj * CUDA_BZ;

                sh_Dist[local_i][local_j] = min(sh_Dist[local_i][local_j],
                                                sh_Dist[local_i][k] + sh_pivot_Dist[k][local_j]);
            }
        }
    }
    
    // Load data back to global memory
    #pragma unroll
    for (int di = 0; di < WORK_PER_THREAD; di++)
    {
        #pragma unroll
        for (int dj = 0; dj < WORK_PER_THREAD; dj++)
        {
            local_i = threadIdx.y + di * CUDA_BZ;
            local_j = threadIdx.x + dj * CUDA_BZ;

            global_i = blockIdx.x * FW_BZ + local_i;
            global_j = pivot_start + local_j;
            d_Dist[global_i*n+global_j] = sh_Dist[local_i][local_j];
        }
    }
}

// Phase 3 kernel: Process all blocks
__global__ void phase3_kernel(int *d_Dist, int n, int round)
{
    // Shared memory for the block
    __shared__ int sh_ik_Dist[FW_BZ][FW_BZ];
    __shared__ int sh_kj_Dist[FW_BZ][FW_BZ];

    int result[WORK_PER_THREAD][WORK_PER_THREAD];
    const int pivot_start = round * FW_BZ;
    int local_i, local_j, global_i, global_j, p_global_j, p_global_i;

    // Load data into shared memory
    #pragma unroll
    for (int di = 0; di < WORK_PER_THREAD; di++)
    {
        #pragma unroll
        for (int dj = 0; dj < WORK_PER_THREAD; dj++)
        {
            local_i = threadIdx.y + di * CUDA_BZ;
            local_j = threadIdx.x + dj * CUDA_BZ;
            p_global_i = pivot_start + local_i;
            p_global_j = pivot_start + local_j;
            global_i = blockIdx.y * FW_BZ + local_i;
            global_j = blockIdx.x * FW_BZ + local_j;

            sh_ik_Dist[local_i][local_j] = d_Dist[global_i*n+p_global_j];
            sh_kj_Dist[local_i][local_j] = d_Dist[p_global_i*n+global_j];
            result[di][dj] = d_Dist[global_i*n+global_j];
        }
    }

    __syncthreads();

    // Process FW_BZ elements for this block
    #pragma unroll
    for (int k = 0; k < FW_BZ; k++)
    {
        #pragma unroll
        for(int di = 0; di < WORK_PER_THREAD; di++)
        {
            #pragma unroll
            for(int dj = 0; dj < WORK_PER_THREAD; dj++)
            {
                local_i = threadIdx.y + di * CUDA_BZ;
                local_j = threadIdx.x + dj * CUDA_BZ;

                result[di][dj] = min(result[di][dj],
                                     sh_ik_Dist[local_i][k] + sh_kj_Dist[k][local_j]);
            }
        }
    }

    // Load data back to global memory
    #pragma unroll
    for (int di = 0; di < WORK_PER_THREAD; di++)
    {
        #pragma unroll
        for (int dj = 0; dj < WORK_PER_THREAD; dj++)
        {
            local_i = threadIdx.y + di * CUDA_BZ;
            local_j = threadIdx.x + dj * CUDA_BZ;
            global_i = blockIdx.y * FW_BZ + local_i;
            global_j = blockIdx.x * FW_BZ + local_j;

            d_Dist[global_i*n+global_j] = result[di][dj];
        }
    }

}

void block_FW()
{
    int* d_Dist;

    hipMalloc(&d_Dist, n*n*sizeof(int));
    hipMemcpy(d_Dist, h_Dist, n*n*sizeof(int), hipMemcpyHostToDevice);

    const int round = n / FW_BZ;
    
    dim3 block(CUDA_BZ, CUDA_BZ);
    
    #pragma unroll
    for (int r = 0; r < round; ++r) {
        phase1_kernel<<<1, block>>>(d_Dist, n, r);
        phase2_kernel_row<<<dim3(round, 1), block>>>(d_Dist, n, r);
        phase2_kernel_col<<<dim3(round, 1), block>>>(d_Dist, n, r);
        phase3_kernel<<<dim3(round, round), block>>>(d_Dist, n, r);
    }
    hipDeviceSynchronize();

    hipMemcpy(h_Dist, d_Dist, n*n*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_Dist);
}

// Main function remains mostly the same for now
int main(int argc, char* argv[])
{
    input(argv[1]);
    block_FW();
    output(argv[2]);
    
    return 0;
}